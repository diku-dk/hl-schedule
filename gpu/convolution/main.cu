#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <limits>

#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

#include "goldenSeq.h"

#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      exit(1);                                                    \
    }                                                             \
  } while(0)
#define CUDA_SAFE_CALL(x)                                         \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      std::cerr << "\nerror: " #x " failed with error "           \
                << msg << '\n';                                   \
      exit(1);                                                    \
    }                                                             \
  } while(0)

int main() {
    const int N = 1;
    const int P = 112;
    const int Q = 112;
    const int K = 64;
    const int C = 3;
    const int R = 7;
    const int S = 7;

    float *images = (float *) malloc(N * (2 * P + R - 1) * (2 * Q + S - 1) * C * sizeof(float));
    for (int i = 0; i < N * (2 * P + R - 1) * (2 * Q + S - 1) * C; ++i) images[i] = (i % 10) + 1;
    float *filter = (float *) malloc(K * R * S * C * sizeof(float));
    for (int i = 0; i < K * R * S * C; ++i) filter[i] = (i % 10) + 1;
    float *out = (float *) malloc(N * P * Q * K * sizeof(float));
    for (int i = 0; i < N * P * Q * K; ++i) out[i] = 0;

    float *out_gold = (float *) malloc(N * P * Q * K * sizeof(float));
    goldenSeq(images, filter, out_gold, N, P, Q, K, C, R, S);

    std::ifstream kernel_stream("kernels.cuda");
    std::string mcc_kernel = std::string(std::istreambuf_iterator<char>(kernel_stream),
                                         std::istreambuf_iterator<char>());
    kernel_stream.close();

    hiprtcProgram prog;
    NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, mcc_kernel.c_str(), "kernels.cuda", 0, nullptr, nullptr));
    hiprtcResult compileResult = hiprtcCompileProgram(prog, 0, nullptr);
    if (compileResult != HIPRTC_SUCCESS) {
        exit(1);
    }
    size_t ptxSize;
    NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
    char *ptx = new char[ptxSize];
    NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));
    NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

    hipDevice_t cuDevice;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    CUDA_SAFE_CALL(hipInit(0));
    CUDA_SAFE_CALL(hipDeviceGet(&cuDevice, 0));
    CUDA_SAFE_CALL(hipCtxCreate(&context, 0, cuDevice));
    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 0, nullptr, nullptr));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, "convolution"));

    hipDeviceptr_t dimages, dfilter, dout;
    CUDA_SAFE_CALL(hipMalloc(&dimages, N * (2 * P + R - 1) * (2 * Q + S - 1) * C * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&dfilter, K * R * S * C * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&dout, N * P * Q * K * sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpyHtoD(dimages, images, N * (2 * P + R - 1) * (2 * Q + S - 1) * C * sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpyHtoD(dfilter, filter, K * R * S * C * sizeof(float)));

    hipEvent_t start, end;
    CUDA_SAFE_CALL(hipEventCreateWithFlags(&start, 0));
    CUDA_SAFE_CALL(hipEventCreateWithFlags(&end, 0));
    void *args[] = {&dimages, &dfilter, &dout};
    for (int warmup = 0; warmup < 10; ++warmup) {
        CUDA_SAFE_CALL(
            hipModuleLaunchKernel(kernel,
                           2 * 7 * 7, 1, 1,
                           16 * 4 * 4, 1, 1,
                           0, nullptr,
                           args, nullptr));
        CUDA_SAFE_CALL(hipCtxSynchronize());
    }
    size_t min_runtime = std::numeric_limits<size_t>::max();
    for (int evaluation = 0; evaluation < 200; ++evaluation) {
        CUDA_SAFE_CALL(hipEventRecord(start, nullptr));
        CUDA_SAFE_CALL(
            hipModuleLaunchKernel(kernel,
                           2 * 7 * 7, 1, 1,
                           16 * 4 * 4, 1, 1,
                           0, nullptr,
                           args, nullptr));
        CUDA_SAFE_CALL(hipEventRecord(end, nullptr));
        CUDA_SAFE_CALL(hipCtxSynchronize());

        float runtime_ms;
        CUDA_SAFE_CALL(hipEventElapsedTime(&runtime_ms, start, end));
        size_t runtime = runtime_ms * 1000000;
        if (runtime < min_runtime)
            min_runtime = runtime;
    }

    CUDA_SAFE_CALL(hipMemcpyDtoH(out, dout, N * P * Q * K * sizeof(float)));
    for (int n = 0; n < N; ++n)
    for (int p = 0; p < P; ++p)
    for (int q = 0; q < Q; ++q)
    for (int k = 0; k < K; ++k)
        if (out[n * P * Q * K + p * Q * K + q * K + k] != out_gold[n * P * Q * K + p * Q * K + q * K + k]) {
            printf("incorrect result at index %d (expected %.0f, actual %.0f)\n",
                   n * P * Q * K + p * Q * K + q * K + k,
                   out_gold[n * P * Q * K + p * Q * K + q * K + k],
                   out[n * P * Q * K + p * Q * K + q * K + k]);
            exit(1);
        }
    printf("result is correct\n");
    printf("runtime: %lu ns\n", min_runtime);

    CUDA_SAFE_CALL(hipFree(dimages));
    CUDA_SAFE_CALL(hipFree(dfilter));
    CUDA_SAFE_CALL(hipFree(dout));
    CUDA_SAFE_CALL(hipModuleUnload(module));
    CUDA_SAFE_CALL(hipCtxDestroy(context));
    delete[] ptx;
    free(images);
    free(filter);
    free(out);
    free(out_gold);
}
